#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>

#include "rvtx/cuda/setup.cuh"
#include "rvtx/optix/texture.cuh"

// Strongly based on
// https://github.com/NVIDIA/cuda-samples/blob/3559ca4d088e12db33d6918621cab5c998ccecf1/Samples/3_CUDA_Features/bindlessTexture/bindlessTexture.cpp#L101
namespace rvtx::optix
{
    static uint32_t getMipMapLevels( uint32_t width, uint32_t height )
    {
        std::size_t sz     = std::max( width, height );
        uint        levels = 0;
        while ( sz != 0 )
        {
            sz /= 2;
            levels++;
        }

        return levels;
    }

    __device__ __inline__ uchar4 to_uchar4( float4 vec )
    {
        return make_uchar4( (uint8_t)vec.x, (uint8_t)vec.y, (uint8_t)vec.z, (uint8_t)vec.w );
    }

    __global__ void d_mipmap( hipSurfaceObject_t mipOutput, hipTextureObject_t mipInput, uint imageW, uint imageH )
    {
        uint x = blockIdx.x * blockDim.x + threadIdx.x;
        uint y = blockIdx.y * blockDim.y + threadIdx.y;

        float px = 1.f / float( imageW );
        float py = 1.f / float( imageH );

        if ( ( x < imageW ) && ( y < imageH ) )
        {
            // take the average of 4 samples

            // we are using the normalized access to make sure non-power-of-two textures
            // behave well when downsized.
            float4 color = ( tex2D<float4>( mipInput, ( x + 0 ) * px, ( y + 0 ) * py ) )
                           + ( tex2D<float4>( mipInput, ( x + 1 ) * px, ( y + 0 ) * py ) )
                           + ( tex2D<float4>( mipInput, ( x + 1 ) * px, ( y + 1 ) * py ) )
                           + ( tex2D<float4>( mipInput, ( x + 0 ) * px, ( y + 1 ) * py ) );

            color /= 4.f;
            surf2Dwrite( color, mipOutput, x * sizeof( float4 ), y );
        }
    }

    void generateMipMaps( hipMipmappedArray_t mipmapArray, hipExtent size )
    {
        size_t width  = size.width;
        size_t height = size.height;
        uint   level  = 0;
        while ( width != 1 || height != 1 )
        {
            width /= 2;
            width = std::max( static_cast<std::size_t>( 1 ), width );
            height /= 2;
            height = std::max( static_cast<std::size_t>( 1 ), height );

            hipArray_t levelFrom;
            cuda::cudaCheck( hipGetMipmappedArrayLevel( &levelFrom, mipmapArray, level ) );
            hipArray_t levelTo;
            cuda::cudaCheck( hipGetMipmappedArrayLevel( &levelTo, mipmapArray, level + 1 ) );

            hipExtent levelToSize = {};
            cuda::cudaCheck( hipArrayGetInfo( nullptr, &levelToSize, nullptr, levelTo ) );
            assert( levelToSize.width == width );
            assert( levelToSize.height == height );
            assert( levelToSize.depth == 0 );

            // generate texture object for reading
            hipTextureObject_t texInput;
            hipResourceDesc    texRes = {};
            texRes.resType             = hipResourceTypeArray;
            texRes.res.array.array     = levelFrom;

            hipTextureDesc texDescr  = {};
            texDescr.normalizedCoords = 1;
            texDescr.filterMode       = hipFilterModeLinear;

            texDescr.addressMode[ 0 ] = hipAddressModeClamp;
            texDescr.addressMode[ 1 ] = hipAddressModeClamp;
            texDescr.addressMode[ 2 ] = hipAddressModeClamp;

            texDescr.readMode = hipReadModeElementType;

            cuda::cudaCheck( hipCreateTextureObject( &texInput, &texRes, &texDescr, NULL ) );

            // generate surface object for writing

            hipSurfaceObject_t surfOutput;
            hipResourceDesc    surfRes = {};
            surfRes.resType             = hipResourceTypeArray;
            surfRes.res.array.array     = levelTo;

            cuda::cudaCheck( hipCreateSurfaceObject( &surfOutput, &surfRes ) );

            // run mipmap kernel
            dim3 blockSize( 16, 16, 1 );
            dim3 gridSize(
                ( (uint)width + blockSize.x - 1 ) / blockSize.x, ( (uint)height + blockSize.y - 1 ) / blockSize.y, 1 );

            d_mipmap<<<gridSize, blockSize>>>( surfOutput, texInput, (uint)width, (uint)height );

            cuda::cudaCheck( hipDeviceSynchronize() );
            cuda::cudaCheck( hipGetLastError() );

            cuda::cudaCheck( hipDestroySurfaceObject( surfOutput ) );
            cuda::cudaCheck( hipDestroyTextureObject( texInput ) );

            level++;
        }
    }

    Texture Texture::From( uint32_t width, uint32_t height, ConstSpan<float> data )
    {
        Texture result {};
        result.m_width  = width;
        result.m_height = height;

        // how many mipmaps we need
        const uint32_t levels = getMipMapLevels( result.m_width, result.m_height );
        result.m_lodLevels      = std::max( 1.f, static_cast<float>( levels - 1 ) );

        // how many mipmaps we need
        hipChannelFormatDesc desc   = hipCreateChannelDesc<float4>();
        hipExtent            extent = { width, height, 0 };
        cuda::cudaCheck( hipMallocMipmappedArray( &result.m_mipmapArray, &desc, extent, levels ) );

        // upload level 0
        hipArray_t level0;
        cuda::cudaCheck( hipGetMipmappedArrayLevel( &level0, result.m_mipmapArray, 0 ) );

        hipMemcpy3DParms copyParams = { 0 };
        copyParams.srcPtr            = make_hipPitchedPtr( (void *)data.ptr, width * sizeof( float4 ), width, height );
        copyParams.dstArray          = level0;
        copyParams.extent            = extent;
        copyParams.extent.depth      = 1;
        copyParams.kind              = hipMemcpyHostToDevice;
        cuda::cudaCheck( hipMemcpy3D( &copyParams ) );

        // compute rest of mipmaps based on level 0
        generateMipMaps( result.m_mipmapArray, extent );

        // generate bindless texture object
        hipResourceDesc resDescr  = {};
        resDescr.resType           = hipResourceTypeMipmappedArray;
        resDescr.res.mipmap.mipmap = result.m_mipmapArray;

        hipTextureDesc texDescr  = {};
        texDescr.normalizedCoords = 1;
        texDescr.filterMode       = hipFilterModeLinear;
        texDescr.mipmapFilterMode = hipFilterModeLinear;

        texDescr.addressMode[ 0 ] = hipAddressModeClamp;
        texDescr.addressMode[ 1 ] = hipAddressModeClamp;
        texDescr.addressMode[ 2 ] = hipAddressModeClamp;

        texDescr.maxMipmapLevelClamp = float( levels - 1 );

        texDescr.readMode = hipReadModeElementType;

        cuda::cudaCheck( hipCreateTextureObject( &result.m_handle, &resDescr, &texDescr, nullptr ) );

        return result;
    }
} // namespace rvtx::optix